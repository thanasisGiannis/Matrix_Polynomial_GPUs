#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cassert>

#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cmath>

/*
	*****************************************************************************************************
	 THIS FILE WASN T WRITTEN BY US, SO WE KNOW NOTHING ABOUT IT, JUST HOW TO CALL THE STRASSEN FUNCTION 
	*****************************************************************************************************
*/

double one = 1.0;
double zero = 0.0;

void GPU_mul(hipblasHandle_t handle, double *A, double *B, double *C,
    int lda, int ldb, int ldc,
    int XA, int XB, int XC,
    int YA, int YB, int YC,
    double alpha, double beta) {
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, XB, YA, XA, &alpha, B, ldb, A, lda, &beta, C, ldc);
}

void GPU_add(hipblasHandle_t handle, double *A, double *B, double *C,
    int lda, int ldb, int ldc,
    int XA, int YA,
    double alpha, double beta) {
  hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, XA, YA, &alpha, A, lda, &beta, B, ldb, C, ldc);
}

void verifyByCUBLAS(hipblasHandle_t handle, double *d_A, double *d_B, double *d_C, int M, int N, int K) {
  double one = 1.0;
  double zero = 0.0;
#if CMAJOR
  cublasDgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, M, N, K, &one, d_A, M, d_B, K, &zero, d_C, M);
#else
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &one, d_B, N, d_A, K, &zero, d_C, N);
#endif
}

/*
  lda, ldb, ldc is the width in actual memory.
  XA, XB, XC is the width for computation.
  A = XA x YA
  B = XB x YB
  C = XC x YC
*/
void strassen(hipblasHandle_t handle, double *A, double *B, double *C,
    int lda, int ldb, int ldc,
    int XA, int XB, int XC,
    int YA, int YB, int YC,
    int depth) {

  int XA2 = XA / 2;
  int XB2 = XB / 2;
  int XC2 = XC / 2;
  
  int YA2 = YA / 2;
  int YB2 = YB / 2;
  int YC2 = YC / 2;

  double *W_1, *W_2;
  int lw1 = (XA2 > XC2 ? XA2 : XC2);
  int lw2 = XB2;
  hipMalloc((void **)&W_1, lw1 * YA2 * sizeof(double));
  hipMalloc((void **)&W_2, lw2 * YB2 * sizeof(double));

  int dXA = XA2;
  int dYA = YA2 * lda;
  int dXB = XB2;
  
  int dYB = YB2 * ldb;
  int dXC = XC2;
  int dYC = YC2 * ldc;

  double *A11, *A12, *A21, *A22;
  double *B11, *B12, *B21, *B22;
  double *C11, *C12, *C21, *C22;
  
  A11 = A;
  A12 = A + dXA;
  A21 = A + dYA;
  A22 = A + dXA + dYA;
  
  B11 = B;
  B12 = B + dXB;
  B21 = B + dYB;
  B22 = B + dXB + dYB;
  
  C11 = C;
  C12 = C + dXC;
  C21 = C + dYC;
  C22 = C + dXC + dYC;

  /* cutoff criteria */
  bool stop = false;
  
#if 0
  int cutoff = 2048;
  float mm = cutoff / XB2;
  float nn = cutoff / YA2;
  float kk = cutoff / XA2;
  if ((mm + nn + kk) >= 3) {
      stop = true;
  }
#endif

  if (depth <= 1 || stop) {
    GPU_add(handle, A11, A21, W_1, lda, lda, lw1, XA2, YA2, 1.0, -1.0); // W_1 = A11 - A21
    GPU_add(handle, B22, B12, W_2, ldb, ldb, lw2, XB2, YB2, 1.0, -1.0); // W_2 = B22 - B12
    GPU_mul(handle, W_1, W_2, C21, lw1, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, 1.0, 0.0); // C21 = W_1 * W_2
    GPU_add(handle, A21, A22, W_1, lda, lda, lw1, XA2, YA2, 1.0,  1.0); // W_1 = A21 + A22
    GPU_add(handle, B12, B11, W_2, ldb, ldb, lw2, XB2, YB2, 1.0, -1.0); // W_2 = B12 - B11
    GPU_mul(handle, W_1, W_2, C22, lw1, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, 1.0, 0.0); // C22 = W_1 * W_2
    GPU_add(handle, W_1, A11, W_1, lw1, lda, lw1, XA2, YA2, 1.0, -1.0); // W_1 = W_1- A11
    GPU_add(handle, B22, W_2, W_2, ldb, lw2, lw2, XB2, YB2, 1.0, -1.0); // W_2 = B22 - W_2
    GPU_mul(handle, W_1, W_2, C11, lw1, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, 1.0, 0.0); // C11 = W_1 * W_2
    GPU_add(handle, A12, W_1, W_1, lda, lw1, lw1, XA2, YA2, 1.0, -1.0); // W_1 = A12 - W_1
    GPU_mul(handle, W_1, B22, C12, lw1, ldb, ldc, XA2, XB2, XC2, YA2, YB2, YC2, 1.0, 0.0); // C12 = W_1 * B22
    GPU_add(handle, C22, C12, C12, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C12 = C22 + C12
    GPU_mul(handle, A11, B11, W_1, lda, ldb, lw1, XA2, XB2, XC2, YA2, YB2, YC2, 1.0, 0.0); // W_1= A11 * B11
    GPU_add(handle, W_1, C11, C11, lw1, ldc, ldc, XC2, YC2, 1.0,  1.0); // C11 = W_1 + C11
    GPU_add(handle, C11, C12, C12, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C12 = C11 + C12
    GPU_add(handle, C11, C21, C11, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C11 = C11 + C21
    GPU_add(handle, W_2, B21, W_2, lw2, ldb, lw2, XB2, YB2, 1.0, -1.0); // W_2 = W_2- B21
    GPU_mul(handle, A22, W_2, C21, lda, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, 1.0, 0.0); // C21 = A22 * W_2
    GPU_add(handle, C11, C21, C21, ldc, ldc, ldc, XC2, YC2, 1.0, -1.0); // C11 = C11 - C21
    GPU_add(handle, C11, C22, C22, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C22 = C11 + C22
    GPU_mul(handle, A12, B21, C11, lda, ldb, ldc, XA2, XB2, XC2, YA2, YB2, YC2, 1.0, 0.0); // C11 = A12 * B21
    GPU_add(handle, W_1, C11, C11, lw1, ldc, ldc, XC2, YC2, 1.0,  1.0); // C11 = W_1+ C11
  }
  else {
    GPU_add(handle, A11, A21, W_1, lda, lda, lw1, XA2, YA2, 1.0, -1.0); // W_1 = A11 - A21
    GPU_add(handle, B22, B12, W_2, ldb, ldb, lw2, XB2, YB2, 1.0, -1.0); // W_2 = B22 - B12
    strassen(handle, W_1, W_2, C21, lw1, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, depth - 1);
    GPU_add(handle, A21, A22, W_1, lda, lda, lw1, XA2, YA2, 1.0,  1.0); // W_1 = A21 + A22
    GPU_add(handle, B12, B11, W_2, ldb, ldb, lw2, XB2, YB2, 1.0, -1.0); // W_2 = B12 - B11
    strassen(handle, W_1, W_2, C22, lw1, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, depth - 1);
    GPU_add(handle, W_1, A11, W_1, lw1, lda, lw1, XA2, YA2, 1.0, -1.0); // W_1 = W_1- A11
    GPU_add(handle, B22, W_2, W_2, ldb, lw2, lw2, XB2, YB2, 1.0, -1.0); // W_2 = B22 - W_2
    strassen(handle, W_1, W_2, C11, lw1, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, depth - 1);
    GPU_add(handle, A12, W_1, W_1, lda, lw1, lw1, XA2, YA2, 1.0, -1.0); // W_1 = A12 - W_1
    strassen(handle, W_1, B22, C12, lw1, ldb, ldc, XA2, XB2, XC2, YA2, YB2, YC2, depth - 1);
    GPU_add(handle, C22, C12, C12, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C12 = C22 + C12
    strassen(handle, A11, B11, W_1, lda, ldb, lw1, XA2, XB2, XC2, YA2, YB2, YC2, depth - 1);
    GPU_add(handle, W_1, C11, C11, lw1, ldc, ldc, XC2, YC2, 1.0,  1.0); // C11 = W_1 + C11
    GPU_add(handle, C11, C12, C12, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C12 = C11 + C12
    GPU_add(handle, C11, C21, C11, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C11 = C11 + C21
    GPU_add(handle, W_2, B21, W_2, lw2, ldb, lw2, XB2, YB2, 1.0, -1.0); // W_2 = W_2- B21
    strassen(handle, A22, W_2, C21, lda, lw2, ldc, XA2, XB2, XC2, YA2, YB2, YC2, depth - 1);
    GPU_add(handle, C11, C21, C21, ldc, ldc, ldc, XC2, YC2, 1.0, -1.0); // C11 = C11 - C21
    GPU_add(handle, C11, C22, C22, ldc, ldc, ldc, XC2, YC2, 1.0,  1.0); // C22 = C11 + C22
    strassen(handle, A12, B21, C11, lda, ldb, ldc, XA2, XB2, XC2, YA2, YB2, YC2, depth - 1);
    GPU_add(handle, W_1, C11, C11, lw1, ldc, ldc, XC2, YC2, 1.0,  1.0); // C11 = W_1+ C11
  }
  hipFree(W_1);
  hipFree(W_2);

  /* dynamic peeling fix-up */
  int pxa = XA % 2;
  int pya = YA % 2;
  int pxb = XB % 2;
  int pyb = YB % 2;
  int pxc = XC % 2;
  int pyc = YC % 2;
  
  int nxa = XA - pxa;
  int nya = YA - pya;
  int nxb = XB - pxb;
  int nyb = YB - pyb;
  int nxc = XC - pxc;
  int nyc = YC - pyc;

  double *a12, *a21;
  double *b12, *b21;
  double *c12, *c21;
  int dxa = nxa;
  int dya = nya * lda;
  int dxb = nxb;
  int dyb = nyb * ldb;
  int dxc = nxc;
  int dyc = nyc * ldc;
  
  a12 = A + dxa;
  a21 = A + dya;
  // a22 = A + dxa + dya;
  b12 = B + dxb;
  b21 = B + dyb;
  // b22 = B + dxb + dyb;
  c12 = C + dxc;
  c21 = C + dyc;
  // c22 = C + dxc + dyc;

  /* 
    A11 = nxa x nya
    a12 = pxa x nya
    a21 = nxa x pya
    a22 = pxa x pya
   */


  GPU_mul(handle, a21, B11, c21, lda, ldb, ldc, nxa,  XB,  XC, pya, nyb, pyc, 1.0, 0.0);
  GPU_mul(handle, A11, b12, c12, lda, ldb, ldc, nxa, pxb, pxc,  YA, nyb,  YC, 1.0, 0.0);
  GPU_mul(handle, a12, b21, C11, lda, ldb, ldc, pxa,  XB,  XC,  YA, pyb,  YC, 1.0, 1.0);
}



void GPU_strassen(hipblasHandle_t handle,double *A, double *B, double *C,
	int lda, int ldb, int ldc,
	int XA, int XB, int XC,
	int YA, int YB, int YC,
	int depth) {

	double *dev_A,*dev_B,*dev_C;
	
	
	if(hipMalloc((void**)&dev_A,XA*YA*sizeof(double))!=hipSuccess){return;}
	if(hipMalloc((void**)&dev_B,XB*YB*sizeof(double))!=hipSuccess){
		hipFree(dev_A);
		return;
	}
	if(hipMalloc((void**)&dev_C,XC*YC*sizeof(double))!=hipSuccess){
		hipFree(dev_A);
		hipFree(dev_B);
		return;
	}
	
	
	hipblasSetMatrix(YA,XA,sizeof(double),A,lda,dev_A,YA);
	
	if(hipblasSetMatrix(YB,XB,sizeof(double),B,ldb,dev_B,YB)!=HIPBLAS_STATUS_SUCCESS){
		printf("ERROOOOOOOOR!\n");
	}
	
	strassen(handle, dev_A,dev_B,dev_C,YA,YB,YC,YA,YB,YC,XA,XB,XC,depth);
	/*double one=1.1;
	double zero=0.0;	
	cublasDgemm(handle,CUBLAS_OP_T,CUBLAS_OP_T,YB,YA,XA,&one,dev_B,ldb,dev_A,lda,&zero,dev_C,ldc);
	*/hipblasGetMatrix(YC,XC,sizeof(double),dev_C,YC,C,ldc);
	
	
	
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	
}




void GPU_MUL_TEST(double *A, double *B, double *C, int x, int y, int z) {

	double *dev_A,*dev_B,*dev_C;
	
	hipblasHandle_t handle;

	if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS){
			printf("Error Initializing Cublas (Strassen Func)");
			return;
	}
		
	
	if(hipMalloc((void**)&dev_A,x*y*sizeof(double))!=hipSuccess){return;}
	if(hipMalloc((void**)&dev_B,y*z*sizeof(double))!=hipSuccess){
		hipFree(dev_A);
		return;
	}
	if(hipMalloc((void**)&dev_C,x*z*sizeof(double))!=hipSuccess){
		hipFree(dev_A);
		hipFree(dev_B);
		return;
	}
	

	
	hipblasSetMatrix(y,x,sizeof(double),A,y,dev_A,y);
	hipblasSetMatrix(z,y,sizeof(double),B,z,dev_B,z);
	
	verifyByCUBLAS(handle, dev_A, dev_B, dev_C, x,  y, z);
	
	hipblasGetMatrix(z,x,sizeof(double),dev_C,z,C,z);
	
	hipblasDestroy(handle);
	
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	
}





